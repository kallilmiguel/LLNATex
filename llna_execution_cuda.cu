#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<dirent.h>
#include<string.h>
#include<stdbool.h>

#include "hip/hip_runtime.h"
#include ""

#define NB_SIZE 8

typedef struct{
    bool rule[NB_SIZE+1];
}rules;

typedef struct node{
    int vertex;
    struct node* next;
}node;

node* createNode(int);

typedef struct{
    int numVertices;
    node** adjLists;
}Graph;

//Create a node
node* createNode(int v){
    node *newNode = (node*) malloc(sizeof(node));
    newNode->vertex = v;
    newNode->next = NULL;
    return newNode;
}

//Create a graph
Graph* createGraph(int vertices){
    Graph *graph = (Graph*) malloc(sizeof(Graph));
    graph->numVertices = vertices;

    graph->adjLists = (node**) malloc(vertices * sizeof(node*));

    int i;
    for(i=0; i<vertices;i++){
        graph->adjLists[i] = NULL;
    }

    return graph;
}

//Add edge (in bidirectional graph)
void addEdge(Graph* graph, int source, int destiny){
    //add edge from s to d
    node* newNode = createNode(destiny);
    newNode->next = graph->adjLists[source];
    graph->adjLists[source] = newNode;

}

//print the graph
void printGraph(Graph* graph) 
{
    int v;
    for (v=0; v<graph->numVertices;v++){
        node* temp = graph->adjLists[v];
        printf("\nVertex %d\n ", v);
        while(temp){
            printf("%d -> ", temp->vertex);
            temp = temp->next;
        }
        printf("\n");
    }
}

Graph* construct_graph_from_image(int rows, int cols, int *img, int R){

    //Create the graph
    Graph *G = createGraph(rows*cols);

    for (int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            for(int y=i-R;y<=i+R;y++){
                if(y >= 0 && y<rows){
                    for(int x=j-R;x<=i+R;x++){
                        if(x >= 0 && x<cols){
                            double d = sqrt(pow(i-y,2)+pow(j-x,2));
                            if(img[j+i*cols] <= img[x+y*cols] && d<=R){
                                addEdge(G, j+i*cols,x+y*cols);
                            }
                        }
                    }
                }
            }
        }
    }
    
    return G;

}

rules* getAllRules(){
    rules *allRules = (rules*) malloc(sizeof(rules)*512);

    int counter=0;

    FILE *ruleFile;
    char *rulePath = "data/rules/rules.csv";

    ruleFile = fopen(rulePath, "r");
    rules *pRules = &allRules[counter];
    while(1){
        char c=fgetc(ruleFile);
        if(c== EOF){
            break;
        }
        else if(c == '\n'){
            counter++;
            pRules = &allRules[counter];
        }
        else if(c != ','&& c != ' '){
            allRules[counter].rule[(int)c-48]=true;
        }
    }
    return allRules;
}

bool isInList(int array[], int value){
    int size = sizeof(array)/sizeof(array[0]);


    for(int i=0;i < size; i++){
        if(array[i]==value)
            return true;
    }
    return false;
}

void generateAllTep(rules* bRules, rules *sRules, Graph *G, int number_of_nodes, int steps){

    int counterB = 0;
    int counterS = 0;
    
    double density[number_of_nodes];
    double resolution[NB_SIZE+1];

    for(int i=0;i<NB_SIZE+1;i++){
        resolution[i] = i+1/(double)NB_SIZE+1;
    }

    FILE *rules = fopen("data/rules/rules.csv", "r");

    while(counterB < 512){
        counterS=0;
        printf("\nBirth Rule number %d\n", counterB);
        while(counterS < 512){
            bool TEP[steps][number_of_nodes];
            for(int i=0;i<number_of_nodes;i++){
                TEP[0][i] = rand() & 1;
            }
            for(int i=1;i<steps;i++){
                for(int j=0;j<number_of_nodes;j++){
                    int degree=0;
                    int num_neighbors_alive=0;
                    node *p = G->adjLists[j];
                    while(p){
                        p = p->next;
                        degree++;
                        if(TEP[i-1][j]==1){
                            num_neighbors_alive+=1;
                        }
                    }
                    density[i] = (double)num_neighbors_alive/(double)degree;
                    if(TEP[i-1][j] == 0){
                        for(int k=0;k<NB_SIZE+1;k++){
                            if(bRules[counterB].rule[k] == true && density[i] >= resolution[k] && density[i] < resolution[k+1]){
                                TEP[i][j]=1;
                                break;
                            }
                            TEP[i][j]=0;
                        }
                    }
                    else{
                        for(int k=0;k<NB_SIZE+1;k++){
                            if(sRules[counterS].rule[k] == true && density[i] >= resolution[k] && density[i] < resolution[k+1]){
                                TEP[i][j]=1;
                                break;
                            }
                            TEP[i][j]=0;
                        }  
                    }
                }
            }
            counterS++;
        }
        counterB++;
    }

}

const char *get_filename_ext(const char *filename){
    const char *dot = strrchr(filename, '.');
    if(!dot || dot == filename) return "";
    return dot + 1;
}

void generateTepGPU(rules* bRules, rules *sRules, Graph *G, int number_of_nodes, int steps){

    int counterB = 0;
    int counterS = 0;
    
    double density[number_of_nodes];
    double resolution[NB_SIZE+1];

    for(int i=0;i<NB_SIZE+1;i++){
        resolution[i] = i+1/(double)NB_SIZE+1;
    }

    dim3 block_size(128);
    dim3 grid_size(8);

    Graph *gpu_graph;
    hipMalloc((void**) &gpu_graph, sizeof(G));
    hipMemcpy(gpu_graph, G, sizeof(G), hipMemcpyHostToDevice);

    FILE *rules = fopen("data/rules/rules.csv", "r");

    while(counterB < 512){
        counterS=0;
        printf("\nBirth Rule number %d\n", counterB);
        while(counterS < 512){
            bool TEP[steps][number_of_nodes];
            hipMalloc((void**) &TEP, sizeof(TEP));

            rule* gpu_bRule;
            hipMalloc((void**)&gpu_bRule, sizeof(rules));
            rule* gpu_sRule;
            hipMalloc((void**)&gpu_sRule, sizeof(rules));

            hipMemcpy(bRules[counterB], gpu_bRule, sizeof(rules),hipMemcpyDeviceToHost);
            hipMemcpy(sRules[counterS], gpu_sRule, sizeof(rules),hipMemcpyDeviceToHost);

            for(int i=0;i<number_of_nodes;i++){
                TEP[0][i] = rand() & 1;
            }
            for(int i=1;i<steps;i++){
                execution_step(TEP, i, number_of_nodes, resolution, gpu_graph, gpu_bRule, gpu_sRule);
            }
            counterS++;
        }
        counterB++;
    }

}

__global__ void execution_step(int** TEP, int iter, int number_of_nodes, double* resolution, 
Graph* G, rules* bRule, rules *sRule){

    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    int degree = 0;
    int num_neighbors_alive=0;
    node *p = G-> adjLists[iter];

    while(p){
        p = p->next;
        degree++;
        if(TEP[i-1][j]==1){
                num_neighbors_alive+=1;
        }
    }
    density[i] = (double)num_neighbors_alive/(double)degree;
    if(TEP[i-1][j] == 0){
        for(int k=0;k<NB_SIZE+1;k++){
            if(bRule.rule[k] == true && density[iter] >= resolution[k] && density[iter] < resolution[k+1]){
                TEP[i][j]=1;
                break;
            }
            TEP[i][j]=0;
        }
    }
    else{
        for(int k=0;k<NB_SIZE+1;k++){
            if(sRules[counterS].rule[k] == true && density[i] >= resolution[k] && density[i] < resolution[k+1]){
                TEP[i][j]=1;
                break;
            }
            TEP[i][j]=0;
        }
    }
å}

int main(void){
    rules *bRules = getAllRules();
    rules *sRules = getAllRules();

    int steps = 350;

    int R=11;
    struct dirent *dir;
    DIR *d;
    char *sdir = (char*) malloc(sizeof(char)*30);
    sprintf(sdir, "data/USPTex/matrices/");
    d = opendir(sdir);
    int counter = 1;
    if(d){
        while((dir = readdir(d)) != NULL){
            if(!strcmp(get_filename_ext(dir->d_name), "txt")){
                printf("Iniciando imagem %d\n", counter);
                FILE *matrix;
                char *path = (char*) malloc(sizeof(char)*30);
                strcpy(path, sdir);
                strcat(path, dir->d_name);
                matrix = fopen(path, "r");

                int value;
                int size;
                 
                fscanf(matrix, "%d", &size);

                int *img = (int*)malloc(sizeof(int)*size);
                int i=0;
                while(fscanf(matrix, "%d", &value)!= EOF){
                    img[i]=value;
                    i++;
                }
                
                int rows = sqrt(size);
                int cols = sqrt(size);

                fclose(matrix);


                Graph *G = construct_graph_from_image(rows, cols, img,R);

                generateAllTep(bRules,sRules, G, rows*cols, steps);

                counter++;

                free(path);
            }
            
        }
    }
    closedir(d);

    return 0;
}


